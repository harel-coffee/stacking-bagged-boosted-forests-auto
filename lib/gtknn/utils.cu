#include "hip/hip_runtime.h"
/*********************************************************************
11	
12	 Copyright (C) 2015 by Wisllay Vitrio
13	
14	 This program is free software; you can redistribute it and/or modify
15	 it under the terms of the GNU General Public License as published by
16	 the Free Software Foundation; either version 2 of the License, or
17	 (at your option) any later version.
18	
19	 This program is distributed in the hope that it will be useful,
20	 but WITHOUT ANY WARRANTY; without even the implied warranty of
21	 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
22	 GNU General Public License for more details.
23	
24	 You should have received a copy of the GNU General Public License
25	 along with this program; if not, write to the Free Software
26	 Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
27	
28	 ********************************************************************/

#include <vector>
#include <string>
#include <sstream>
#include <iostream>
#include <sys/time.h>

#include "utils.cuh"
#include <cudpp.h>

CUDPPHandle theCudpp;

int WARP_SIZE = 32;

void initCudpp() {
    cudppCreate(&theCudpp);
}

std::vector<std::string> &split(const std::string &s, char delim, std::vector<std::string> &elems) {
    std::stringstream ss(s);
    std::string item;
    while (std::getline(ss, item, delim)) {
        elems.push_back(item);
    }
    return elems;
}

std::vector<std::string> split(const std::string &s, char delim) {
    std::vector<std::string> elems;
    split(s, delim, elems);
    return elems;
}


double gettime() {
    timespec ts;
    clock_gettime(CLOCK_REALTIME, &ts);
    return double(ts.tv_sec) + double(ts.tv_nsec) / 1e9;
}

void get_grid_config(dim3 &grid, dim3 &threads) {
    //get the device properties
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    //Adjust the grid dimensions based on the device properties
    int num_blocks = 2 * devProp.multiProcessorCount;
    grid = dim3(num_blocks);
    threads = dim3(devProp.maxThreadsPerBlock / 2);
}

void __gpuAssert(hipError_t stat, int line, char *file) {
    if(stat != hipSuccess) {
        fprintf(stderr, "Error %s at line %d in file %s\n",
                hipGetErrorString(stat), line, file);
        exit(1);
    }
}

//__device__ float atomicAdd(float* address, float val)
//{
//    unsigned long long int* address_as_ull =
//                          (unsigned long long int*)address;
//    unsigned long long int old = *address_as_ull, assumed;
//    do {
//        assumed = old;
//            old = atomicCAS(address_as_ull, assumed,__float_as_longlong(val +
//                               __longlong_as_float(assumed)));
//    } while (assumed != old);
//    return __longlong_as_float(old);
//}
